#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <chrono>

#define CUDA_CHECK_RETURN(value) { \
    hipError_t _m_cudaStat = value; \
    if (_m_cudaStat != hipSuccess) { \
        fprintf(stderr, "Ошибка %s в строке %d в файле %s\n", \
        hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
        exit(1); \
    }}


void fill_random(half *vec, int size) {
    for (int i = 0; i < size; i++) {
        vec[i] = __float2half((float) rand() / RAND_MAX);
    }
}

void mm_half(
    half *A, 
    half *B,
    half *C,
    int M,
    int N,
    int K,
    bool use_tensor_cores
) {

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    const half alpha =  __float2half(1.0);
    const half beta = __float2half(0.0);

    hipblasGemmAlgo_t algo = use_tensor_cores
    ? CUBLAS_GEMM_DEFAULT_TENSOR_OP
    : HIPBLAS_GEMM_DEFAULT;

    hipblasGemmEx(
        cublas_handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K,                             
        &alpha,
        B, HIP_R_16F, N,                   
        A, HIP_R_16F, K,                   
        &beta,
        C, HIP_R_16F, N,                   
        HIP_R_16F,                         
        algo                 
    );

    hipblasDestroy(cublas_handle);
}


void process_mm(int M, int N, int K) {
    int a_size = M * K;  
    int b_size = K * N;
    int c_size = M * N;

    half *h_a, *h_b, *h_c;

    h_a = new half[a_size];
    h_b = new half[b_size];
    h_c = new half[c_size];

    fill_random(h_a, a_size);
    fill_random(h_b, b_size);

    half *d_a, *d_b, *d_c;
    CUDA_CHECK_RETURN(hipMalloc(&d_a, a_size * sizeof(half)));
    CUDA_CHECK_RETURN(hipMalloc(&d_b, b_size * sizeof(half)));
    CUDA_CHECK_RETURN(hipMalloc(&d_c, c_size * sizeof(half)));
    CUDA_CHECK_RETURN(hipMemset(d_c, 0, c_size * sizeof(half)));

    CUDA_CHECK_RETURN(hipMemcpy(d_a, h_a, a_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_b, h_b, b_size * sizeof(half), hipMemcpyHostToDevice));

    auto start = std::chrono::high_resolution_clock::now();
    mm_half(d_a, d_b, d_c, M, N, K, false);
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> mm_no_tc_time = stop - start;
    CUDA_CHECK_RETURN(hipMemcpy(h_c, d_c, c_size * sizeof(half), hipMemcpyDeviceToHost));

    start = std::chrono::high_resolution_clock::now();
    mm_half(d_a, d_b, d_c, M, N, K, true);
    stop = std::chrono::high_resolution_clock::now();   
    std::chrono::duration<float, std::milli> mm_tc_time = stop - start;
    CUDA_CHECK_RETURN(hipMemcpy(h_c, d_c, c_size * sizeof(half), hipMemcpyDeviceToHost));

    printf("Время выполнения cuBLAS FLOAT16 (no tensor cores): %.4f мс\n", mm_no_tc_time.count());
    printf("Время выполнения cuBLAS FLOAT16 (tensor cores): %.4f мс\n\n", mm_tc_time.count());

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main(int argc, char *argv[]) {
    
    int sizes[][3] = {
        {16, 16, 16},
        {32, 32, 32},
        {64, 64, 64},
        {128, 128, 128},
        {256, 256, 256},
        {512, 512, 512},
        {1024, 1024, 1024},
        {2048, 2048, 2048},
        {4096, 4096, 4096},
        {8192, 8192, 8192},
    };

    for (auto& size : sizes) {
        printf("Тестирование для M = %d, N = %d, K = %d\n", size[0], size[1], size[2]);
        process_mm(size[0], size[1], size[2]);
    }

    return 0;
}