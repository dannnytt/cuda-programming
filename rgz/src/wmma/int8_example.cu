#include <hip/hip_runtime.h>
#include <mma.h>
#include <iostream>
#include <chrono>

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define CUDA_CHECK_RETURN(value) { \
    hipError_t _m_cudaStat = value; \
    if (_m_cudaStat != hipSuccess) { \
        fprintf(stderr, "Ошибка %s в строке %d в файле %s\n", \
        hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
        exit(1); \
    }}

void fill_random(int8_t *vec, int size) {
    for (int i = 0; i < size; i++) {
        vec[i] = rand() % 256 - 128;
    }
}

__global__ void wmma_gemm_kernel_int8(int8_t *a, int8_t *b, int32_t *c, int m, int n, int k) {
    using namespace nvcuda;

    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    if (warpM * WMMA_M >= m || warpN * WMMA_N >= n) return;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, int8_t, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, int8_t, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, int32_t> c_frag;

    wmma::fill_fragment(c_frag, 0);

    for (int i = 0; i < k; i += WMMA_K) {
        int aRow = warpM * WMMA_M;
        int aCol = i;
        int bRow = i;
        int bCol = warpN * WMMA_N;

        const int8_t *tile_a = a + aRow * k + aCol;
        const int8_t *tile_b = b + bRow * n + bCol;

        wmma::load_matrix_sync(a_frag, tile_a, k);
        wmma::load_matrix_sync(b_frag, tile_b, n);

        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    int *tile_c = c + warpM * WMMA_M * n + warpN * WMMA_N;
    wmma::store_matrix_sync(tile_c, c_frag, n, wmma::mem_row_major);
}

void process_mm_int8(int M, int N, int K) {
    int a_size =  M * K;
    int b_size =  K * N;
    int c_size =  M * N;

    int8_t *h_a, *h_b;
    int32_t *h_c;
    h_a = new int8_t[a_size];
    h_b = new int8_t[b_size];
    h_c = new int32_t[c_size];

    fill_random(h_a, a_size);
    fill_random(h_b, b_size);

    int8_t *d_a, *d_b;
    int *d_c;
    CUDA_CHECK_RETURN(hipMalloc(&d_a, a_size * sizeof(int8_t)));
    CUDA_CHECK_RETURN(hipMalloc(&d_b, b_size * sizeof(int8_t)));
    CUDA_CHECK_RETURN(hipMalloc(&d_c, c_size * sizeof(int32_t)));
    CUDA_CHECK_RETURN(hipMemset(d_c, 0, c_size * sizeof(int32_t)));

    CUDA_CHECK_RETURN(hipMemcpy(d_a, h_a, a_size * sizeof(int8_t), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_b, h_b, b_size * sizeof(int8_t), hipMemcpyHostToDevice));

    dim3 threads_per_block(16, 16);
    dim3 num_blocks(
        (M + (WMMA_M * 4 - 1)) / (WMMA_M * 4), 
        (N + WMMA_N - 1) / WMMA_N
    );

    auto start = std::chrono::high_resolution_clock::now();
    wmma_gemm_kernel_int8<<<num_blocks, threads_per_block>>>(d_a, d_b, d_c, M, N, K);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<float, std::milli> duration = end - start;
    printf("Время выполнения WMMA для INT8: %.4f\n\n", duration.count());

    hipMemcpy(h_c, d_c, c_size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
}

int main() {
    
    int sizes[][3] = {
        {16, 16, 16},
        {32, 32, 32},
        {64, 64, 64},
        {128, 128, 128},
        {256, 256, 256},
        {512, 512, 512},
        {1024, 1024, 1024},
        {2048, 2048, 2048},
        {4096, 4096, 4096},
        {8192, 8192, 8192},
        {16384, 16384, 16384},
    };

    for (auto& size : sizes) {
        printf("Тестирование WMMA для INT8: M = %d, N = %d, K = %d\n", size[0], size[1], size[2]);
        process_mm_int8(size[0], size[1], size[2]);
    }

    return 0;
}
