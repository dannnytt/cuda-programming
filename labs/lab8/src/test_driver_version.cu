#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int driverVersion;
    hipDriverGetVersion(&driverVersion);
    std::cout << "CUDA Driver Version: " << driverVersion << std::endl;
    return 0;
}