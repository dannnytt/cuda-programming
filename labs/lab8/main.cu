#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include<random>

#define CUDA_CHECK_CALL(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            const char* errStr; \
            hipDrvGetErrorString(err, &errStr); \
            std::cerr << "CUDA error: " << errStr << " at line " << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while (0)

#define VECTOR_SIZE 10


void initialize_random_vectors(std::vector<double> &vector1, std::vector<double> &vector2)  {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dist(0.0, 100.0);

    for (int i = 0; i < VECTOR_SIZE; i++) {
        vector1[i] = dist(gen);
        vector2[i] = dist(gen);
    }
}

int main() {
    
    CUDA_CHECK_CALL(hipInit(0));
    
    hipDevice_t device;
    CUDA_CHECK_CALL(hipDeviceGet(&device, 0));

    hipCtx_t context;
    CUDA_CHECK_CALL(hipCtxCreate(&context, 0, device));

    hipModule_t module;
    CUDA_CHECK_CALL(hipModuleLoad(&module, "ptx/kernel.ptx"));

    hipFunction_t kernel;
    CUDA_CHECK_CALL(hipModuleGetFunction(&kernel, module, "kernel"));

    std::vector<double> h_vector1(VECTOR_SIZE, 5);
    std::vector<double> h_vector2(VECTOR_SIZE, 10);
    std::vector<double> h_result(VECTOR_SIZE, 0);

    initialize_random_vectors(h_vector1, h_vector2);

    hipDeviceptr_t d_vector1, d_vector2, d_result;
    CUDA_CHECK_CALL(hipMalloc(&d_vector1, VECTOR_SIZE * sizeof(double)));
    CUDA_CHECK_CALL(hipMalloc(&d_vector2, VECTOR_SIZE * sizeof(double)));
    CUDA_CHECK_CALL(hipMalloc(&d_result, VECTOR_SIZE * sizeof(double)));

    CUDA_CHECK_CALL(hipMemcpyHtoD(d_vector1, h_vector1.data(), VECTOR_SIZE * sizeof(double)));
    CUDA_CHECK_CALL(hipMemcpyHtoD(d_vector2, h_vector2.data(), VECTOR_SIZE * sizeof(double)));

    int size = VECTOR_SIZE;
    void *args[] = { &d_vector1, &d_vector2, &d_result, &size };
    CUDA_CHECK_CALL(hipModuleLaunchKernel(kernel, 1, 1, 1, VECTOR_SIZE, 1, 1, 0, 0, args, 0));

    CUDA_CHECK_CALL(hipMemcpyDtoH(h_result.data(), d_result, VECTOR_SIZE * sizeof(double)));

    std::cout << "Результат: " << std::endl;
    for (double val : h_result) 
        printf("%-8.2f", val);
    std::cout << std::endl << std::endl;
    
    std::cout << "Cумма: " << std::accumulate(h_result.begin(), h_result.end(), 0) << std::endl;


    CUDA_CHECK_CALL(hipFree(d_vector1));
    CUDA_CHECK_CALL(hipFree(d_vector2));
    CUDA_CHECK_CALL(hipFree(d_result));
    CUDA_CHECK_CALL(hipModuleUnload(module));
    CUDA_CHECK_CALL(hipCtxDestroy(context));

    return 0;
}